
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void get_hello(int *i) {
    i[0] = 123;
}

int main(int argc, char *argv[]) {
    int *buf_device;
    int *buf_host;

    buf_host = (int*)calloc(sizeof(int), 1);
    hipMalloc(&buf_device, sizeof(int));

    hipMemcpy(buf_device, buf_host, sizeof(int), hipMemcpyHostToDevice);
    get_hello<<<1, 1>>>(buf_device);

    hipMemcpy(buf_host, buf_device, sizeof(int), hipMemcpyDeviceToHost);
    printf("Ergebnis: %d\n", buf_host[0]);

    hipFree(&buf_device);
    free(buf_host);    
    return 0;
}